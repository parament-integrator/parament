class GPURunner
{
    public:
        GPURunner(); // Constructor
        ~GPURunner(); // Destructor
        void equiprop(hipComplex *carr, float dt, int pts, hipComplex* out);
        void equiprop2(hipComplex *carr, float dt, int pts, hipComplex* out);
        void set_hamiltonian(hipComplex *H0_host, hipComplex *H1_host, int dim_in);
        void readback();
    private:
        // Handles
        hipblasHandle_t handle;

        // GPU Arrays and constants
        hipComplex* H0;
        hipComplex* H1;
        hipComplex* one_GPU;
        hipComplex *one_GPU_diag;

        // Dimension of the Hilbert space
        int dim;

        // Currently initialized time steps
        int curr_max_pts;

        // Point arrays
        hipComplex* c0;
        hipComplex* c1;
        hipComplex* X;
        hipComplex* D0;
        hipComplex* D1;

        // Commonly used constants
        hipComplex zero = make_hipComplex(0,0);
        hipComplex one = make_hipComplex(1,0);
        hipComplex two = make_hipComplex(2,0);
        hipComplex mone = make_hipComplex(-1,0);
        hipComplex mtwo = make_hipComplex(-2,0);

        // Helper functions
        void diagonal_add(hipComplex num, hipComplex *C_GPU, int batch_size);
        void expmBatched(hipComplex *M, int pts, hipComplex* tmp_out);

        // check vars
        bool hamiltonian_is_set = false;

        // BESSEL COEFFICIENTS
        hipComplex J[MMAX+1];
        float alpha;
        float beta;

        // Device 
        int numSMs;
};

GPURunner::GPURunner()
{
   
    hipblasHandle_t* new_handle;
    new_handle = &handle;
    cublasErrchk(hipblasCreate(new_handle));

    gpuErrchk(hipMalloc(&one_GPU, sizeof(hipComplex)));
    gpuErrchk(hipMemcpy(one_GPU, &one, sizeof(hipComplex), hipMemcpyHostToDevice));

    // BESSEL COEFFICIENTS
    alpha = -2.0;
    beta = 2.0;
    J_arr(J, MMAX, 2.0);

    // No points yet allocated
    curr_max_pts = -1;
    cout << curr_max_pts << endl;

    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    
}

GPURunner::~GPURunner()
{    
   
    cublasErrchk(hipblasDestroy(handle));

    gpuErrchk(hipFree(one_GPU));
    
    if (hamiltonian_is_set == true) {
    gpuErrchk(hipFree(H0));
    gpuErrchk(hipFree(H1));
    gpuErrchk(hipFree(one_GPU_diag));
    }

    if (curr_max_pts > 0){
        hipFree(c0);
        hipFree(c1);    
        hipFree(X);    
        hipFree(D0);
        hipFree(D1);
        
    }
    
    
    std::cout << "Objected destroyed" << std::endl;
}

void GPURunner::set_hamiltonian(hipComplex *H0_host, hipComplex *H1_host, int dim_in)
{
    dim = dim_in; 

    // Allocate GPU memory
    hipMalloc(&H0, dim * dim * sizeof(hipComplex));
    hipMalloc(&H1, dim * dim * sizeof(hipComplex));

    // Transfer to GPU
    hipMemcpy(H0, H0_host, dim * dim * sizeof(hipComplex), hipMemcpyHostToDevice);
    hipMemcpy(H1, H1_host, dim * dim * sizeof(hipComplex), hipMemcpyHostToDevice);

    // Helper Arrays
    hipMalloc(&one_GPU_diag, dim * sizeof(hipComplex));
    hipblasCaxpy(handle, dim, &one, one_GPU, 0, one_GPU_diag, 1);

    hamiltonian_is_set = true;
    nvtxMarkA("Set Hamiltonian routine completed");
}

void GPURunner::diagonal_add(hipComplex num, hipComplex *C_GPU, int batch_size)
{
    
    /*return hipblasCgemmStridedBatched(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                1, dim, 1,
                num,
                one_GPU, 1, 0,
                one_GPU_diag, 1, 0,
                &one,
                C_GPU, dim+1, dim*dim,
                batch_size);    */

    /*return hipblasGemmStridedBatchedEx(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        1, dim, 1,
        num,
        one_GPU, HIP_C_32F, 1, 0,
        one_GPU_diag, HIP_C_32F, 1, 0,
        &one,
        C_GPU, HIP_C_32F, dim+1, dim*dim,
        batch_size, HIP_C_32F,
        CUBLAS_GEMM_ALGO0);*/
    /*for (int i=0;i<dim;i++){
        hipblasCaxpy(handle, batch_size,
            num,
            one_GPU, 0,
            C_GPU+i*dim, dim*dim);
    }*/

    saxpy<<<32*numSMs, 256>>>(num, C_GPU, dim, batch_size);
    return;

}

void GPURunner::readback()
{
    hipComplex* hostprobe = (hipComplex*)malloc(dim * dim * sizeof(hipComplex));
    hipMemcpy(hostprobe, H0, dim * dim * sizeof(hipComplex), hipMemcpyDeviceToHost);
    cout << "H0" << endl;
    printcomplex(hostprobe, dim*dim);

    cout << "H1" << endl;
    hipMemcpy(hostprobe, H1, dim * dim * sizeof(hipComplex), hipMemcpyDeviceToHost);
    printcomplex(hostprobe, dim*dim);

    free(hostprobe);
}

void GPURunner::equiprop(hipComplex *carr, float dt, int pts, hipComplex *out)
{
    // ///////////////////////////////////////////////////////
    // TRANSFER
    // //////////////////////////////////////////////////////
    // Allocate memory for c arrays if needed
    if (curr_max_pts < pts) {
        if (curr_max_pts > 0){
            cout << "Need to free c arrays" <<endl;
            hipFree(&c0);
            hipFree(&c1); 
            hipFree(&X);
        }
        cout << "Need to malloc c arrays" <<endl;
        gpuErrchk(hipMalloc(&c0, pts * sizeof(hipComplex)));
        gpuErrchk(hipMalloc(&c1, pts * sizeof(hipComplex)));
        gpuErrchk(hipMalloc(&X, dim * dim * pts * sizeof(hipComplex)));
        gpuErrchk(hipMalloc(&D0, dim * dim * pts * sizeof(hipComplex)));
        gpuErrchk(hipMalloc(&D1, dim * dim * pts * sizeof(hipComplex)));

        // Memorize how many pts are initalized
        curr_max_pts = pts;
        
        // Fill c0 array with ones
        hipblasCscal(handle, pts, &zero, c0, 1);
        cublasErrchk(hipblasCaxpy(handle, pts, &one, one_GPU, 0, c0, 1));
    }

    // Transfer c1
    hipMemcpy(c1, carr, pts * sizeof(hipComplex), hipMemcpyHostToDevice);

    // ///////////////////////////////////////////////////////
    // EXPAND 
    // ///////////////////////////////////////////////////////
    //hipblasCscal(handle,  dim * dim * pts, &zero, X, 1);
    
    cublasErrchk(hipblasCgemm(handle,
         HIPBLAS_OP_N, HIPBLAS_OP_N,
         dim*dim, pts, 1,
         &one,
         H0, dim*dim,
         c0, 1,
         &zero,
         X, dim*dim)); 
    /*hipblasCgemmStridedBatched(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        dim*dim, 1, 1,
        &one,
        H0, dim*dim,0,
        c0, 1,0,
        &zero,
        X, dim*dim,dim*dim,
        pts);*/

    cublasErrchk(hipblasCgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        dim*dim, pts, 1,
        &one,
        H1, dim*dim,
        c1, 1,
        &one,
        X, dim*dim));



            
    // ///////////////////////////////////////////////////////
    // PROPAGATE
    // ///////////////////////////////////////////////////////
    // Rescale dt
    dt = dt*2/(beta-alpha)*2;
    hipComplex dt_complex;
    dt_complex = make_hipComplex(dt,0);

    //printcomplex(&dt_complex,1);
    
    // Loop
    int k = 0;
    //hipComplex ak;

    hipComplex* ptr_accumulate;

    for (k=MMAX; k >= 0; k--) {
        if (k == MMAX){
            cublasErrchk(hipblasCscal(handle, pts*dim*dim, &zero, D0, 1));
        } 
        else {
        // D0 = D0 + 2 X @ D1 * dt
        cublasErrchk(hipblasCgemmStridedBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_T,
            dim, dim, dim,
            &dt_complex,
            X, dim, dim*dim,
            D1, dim, dim*dim,
            &mone,
            D0, dim, dim*dim,
            pts)) 
        }
        
        // D0 = D0 + I*ak
        diagonal_add(J[k], D0, pts);


        
        // Next step
        k--;
        //cout << k << endl;


        if (k == MMAX-1) {
            ptr_accumulate = &zero;
            //hipblasCscal(handle, pts*dim*dim, &zero, D1, 1);
        }         
        if (k == 0){
            ptr_accumulate = &mtwo;
        }

        // D1 = D1 + 2 X @ D0
        cublasErrchk(hipblasCgemmStridedBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_T,
            dim, dim, dim,
            &dt_complex,
            X, dim, dim*dim,
            D0, dim, dim*dim,
            ptr_accumulate,
            D1, dim, dim*dim,
            pts)); 
        
                
        // Code to test the arrays

        /*int transferpts = pts*dim*dim;
        cout << "---------- D1";
        cout << " -----------" << endl;
        hipComplex* hostprobe = (hipComplex*)malloc(transferpts * sizeof(hipComplex));
        hipMemcpy(hostprobe, D1, transferpts* sizeof(hipComplex), hipMemcpyDeviceToHost);
        printcomplex(hostprobe, transferpts);
        free(hostprobe); */

       // D1 = D1 + I*ak'
       diagonal_add(J[k], D1, pts);

       if (k == MMAX - 1){
           ptr_accumulate = &mone;
       }

    } 
    // D1 contains now the matrix exponentials



    // ///////////////////////////////////////////////////////
    // REDUCE
    // ///////////////////////////////////////////////////////
    // Reduction operation:
    int remain_pts = pts;
    int pad = 0;
    while (remain_pts > 1){

        pad = remain_pts % 2;
        remain_pts = remain_pts/2;

        hipblasCgemmStridedBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_T,
            dim, dim, dim,
            &one,
            D1          , dim, dim*dim*2,
            D1 + dim*dim, dim, dim*dim*2,
            &zero,
            D1, dim, dim*dim,
            remain_pts);
        if (pad>0){
            // One left over, need to copy to Array
            hipblasCcopy(handle, 
                dim*dim,
                D1 + dim*dim*(remain_pts*2), 1,
                D1 + dim*dim*(remain_pts), 1);
            remain_pts += 1;            
        }


    }
    
    // ///////////////////////////////////////////////////////
    // TRANSFER BACK
    // ///////////////////////////////////////////////////////
    hipMemcpy(out, D1, dim * dim  * sizeof(hipComplex), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    
    return;
   
}


void GPURunner::equiprop2(hipComplex *carr, float dt, int pts, hipComplex *out)
{
    hipComplex* c0;
    hipComplex* c1;
    hipMalloc(&c0, pts * sizeof(hipComplex));
    hipMalloc(&c1, pts * sizeof(hipComplex));

    // Rescale dt
    float beta = 2;
    float alpha = -2;
    dt = dt*2/(beta-alpha);
    hipComplex dt_complex;
    dt_complex = make_hipComplex(dt,0);
    
    // Set c0 
    hipblasCscal(handle, pts, &zero, c0, 1);
    hipblasCaxpy(handle, pts, &dt_complex, one_GPU, 0, c0, 1);

    // Transfer c1 & scale by dt
    hipMemcpy(c1, carr, pts * sizeof(hipComplex), hipMemcpyHostToDevice);
    hipblasCsscal(handle, pts, &dt, c1, 1);

    // Initialize iteration arrays
    hipComplex* D0;
    hipMalloc(&D0, dim * dim * pts * sizeof(hipComplex));
    hipComplex* D1;
    hipMalloc(&D1, dim * dim * pts * sizeof(hipComplex));

    // Ensure arrays are zeroed
    hipblasCscal(handle, pts*dim*dim, &zero, D0, 1);
    hipblasCscal(handle, pts*dim*dim, &zero, D1, 1);
    

    // Loop
    int k = 0;
    hipComplex ak;

    //Malloc streams
    // Create a stream for every time step
    hipStream_t *streams = (hipStream_t *) malloc(pts*sizeof(hipStream_t));
    int i;
    for(i=0; i<pts; i++){
        hipStreamCreate(&streams[i]);
    }

    for (i=0;i<pts;i++){
        // set Cublas stream
        hipblasSetStream(handle, streams[i]);
        for (k=MMAX; k >= 0; k--) {
            // D0 = D0 + I*ak
            hipblasCgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                1, dim, 1,
                &ak,
                one_GPU, 1,
                one_GPU_diag, 1,
                &one,
                D0+i*dim*dim, dim+1);    
            // D0 = D0 + 2*X0@D1
            // D0 = D0 + c*X1@D1


            k--;
            // D1 = D1 + I*ak'
            // D1 = D1 + 2*X0@D0
            // D1 = D1 + c*X1@D0

        } 
}

 
    for(i=0; i<pts; i++){
        hipStreamDestroy(streams[i]);
    }

    hipFree(c0);
    hipFree(c1);
    hipFree(D0);
    hipFree(D1);
    
    return;
}



void destroy_GPURunner(GPURunner *pObj){
    delete pObj;
    pObj = NULL;
}

extern "C"
{
    __declspec(dllexport) GPURunner* GPURunner_new() {return new GPURunner();}
    __declspec(dllexport) void GPURunner_del(GPURunner* cls) {destroy_GPURunner(cls);}
    __declspec(dllexport) void GPURunner_readback(GPURunner* cls) {cls->readback();}
    __declspec(dllexport) void GPURunner_sethamiltonian(GPURunner* cls, hipComplex *H0, hipComplex *H1, int dim){cls->set_hamiltonian(H0, H1, dim);}
    __declspec(dllexport) void GPURunner_equiprop(GPURunner* cls, hipComplex *carr, float dt, int pts, hipComplex* out){cls->equiprop(carr, dt, pts, out);}
}