#include "hip/hip_runtime.h"
/* Copyright 2021 Konstantin Herb, Pol Welter. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/


#include <hipblas.h>

#include "diagonal_add.h"

// https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
__global__ void Caxpy_batched(hipComplex a, hipComplex *y, int s, int n)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < n*s; 
         i += blockDim.x * gridDim.x) 
      {
          // dim*(n-floor(n/dim)) + n for indices
          y[s*(i-i/s)+i] = hipCaddf(a,y[s*(i-i/s)+i]);
      }
}

__global__ void Zaxpy_batched(hipDoubleComplex a, hipDoubleComplex *y, int s, int n)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < n*s; 
         i += blockDim.x * gridDim.x) 
      {
          // dim*(n-floor(n/dim)) + n for indices
          y[s*(i-i/s)+i] = hipCadd(a,y[s*(i-i/s)+i]);
      }
}

void diagonal_add(hipComplex num, hipComplex *C_GPU, int batch_size, unsigned int numSMs, unsigned int dim)
{
    Caxpy_batched<<<32*numSMs, 256>>>(num, C_GPU, dim, batch_size);
}

void diagonal_add(hipDoubleComplex num, hipDoubleComplex *C_GPU, int batch_size, unsigned int numSMs, unsigned int dim)
{
    Zaxpy_batched<<<32*numSMs, 256>>>(num, C_GPU, dim, batch_size);
}
