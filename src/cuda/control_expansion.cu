#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdlib.h>
#include <stdio.h>
#define NO_CUDA_STUBS

#include "control_expansion.h"

// 3D thread block indexing
__global__ void generate_magnus(hipComplex *coeffs_in, hipComplex *coeffs_out, int amps, int n, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Amp index
    int k = blockIdx.z * blockDim.z + threadIdx.z;  //Second amp index
    int len_new = (n-3)/2+1;
    hipComplex four = make_hipComplex(4,0);
    hipComplex six = make_hipComplex(6,0);
    
    // commiefactor = i*dt/12
    hipComplex commiefactor = hipCdivf(make_hipComplex(0,1),make_hipComplex(12,0));
    commiefactor = hipCmulf(make_hipComplex(dt,0),commiefactor);
    
    if (i < (n-3)/2+1 && j < amps && k == 0){
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_in[2*i+j*n], hipCmulf(four,coeffs_in[2*i+j*n+1]));
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_out[i+j*len_new], coeffs_in[2*i+j*n+2]);
        coeffs_out[i+j*len_new] = hipCdivf(coeffs_out[i+j*len_new], six);

        // Coefficients for commutator [H0,control H]
        int idx_comm = i+(j+amps)*len_new;
        coeffs_out[idx_comm] = hipCsubf(coeffs_in[2*i+j*n+2],coeffs_in[2*i+j*n]);
        coeffs_out[idx_comm] = hipCmulf(coeffs_out[idx_comm],commiefactor);
        
    

    }


    /*
    // Coefficient calculations involving only one control Hamiltonian
    if (i < len_new && j < amps && k==0){

        // Coefficient reduction for Simpson rule
        int idx_new = i+j*len_new;
        int idx_old = 2*i+n*j;
        //coeff_out[idx_new] = 1/6*coeff_in[idx_old] + 4/6*coeff_in[idx_old+1] + 1/6*coeff_in[idx_old+2];
        coeffs_out[idx_new] = hipCaddf(coeffs_in[idx_old], hipCmulf(four,coeffs_in[idx_old+1]));
        coeffs_out[idx_new] = hipCaddf(coeffs_in[idx_new], coeffs_in[idx_old+2]);
        coeffs_out[idx_new] = hipCdivf(coeffs_in[idx_new], six);

    }
    */

    
    // Coefficient calculations for pairwise commutators of control Hamiltonians
    if (i < len_new && j < k && k < amps){
        int idx_old_amp1 = 2*i+n*j;
        int idx_old_amp2 = 2*i+n*k;
        int idx_new_pair = i+j*len_new+(k-1)*len_new + len_new*amps*2;
        //coeff_out[idx_new_pair] = coeff_in[idx_old_amp1]*coeff_in[idx_old_amp2+2]-coeff_in[idx_old_amp1+2]*coeff_in[idx_old_amp2];
        coeffs_out[idx_new_pair] = hipCmulf(coeffs_in[idx_old_amp1],coeffs_in[idx_old_amp2+2]);
        coeffs_out[idx_new_pair] = hipCsubf(coeffs_out[idx_new_pair],hipCmulf(coeffs_in[idx_old_amp1+2],coeffs_in[idx_old_amp2]));
        coeffs_out[idx_new_pair] = hipCmulf(coeffs_out[idx_new_pair],commiefactor);
        

    }
    

}

// 2D thread block indexing
__global__ void generate_midpoint(hipComplex *coeffs_in, hipComplex *coeffs_out, int amps, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Control field index 
    hipComplex half = make_hipComplex(0.5,0);

    if (i < n - 1 && j < amps){
        coeffs_out[i+j*n] = hipCaddf(coeffs_in[i+j*n],coeffs_in[i+j*n+1]);
        coeffs_out[i+j*n] = hipCmulf(half,coeffs_out[i+j*n]);
    }
}

// 2D thread block indexing
__global__ void generate_simpson(hipComplex *coeffs_in, hipComplex *coeffs_out, int amps, int n)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Control field index 
    hipComplex four = make_hipComplex(4,0);
    hipComplex six = make_hipComplex(6,0);
    int len_new = (n-3)/2+1;

    if (i < (n-3)/2+1 && j < amps){
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_in[2*i+j*n], hipCmulf(four,coeffs_in[2*i+j*n+1]));
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_out[i+j*len_new], coeffs_in[2*i+j*n+2]);
        coeffs_out[i+j*len_new] = hipCdivf(coeffs_out[i+j*len_new], six);
    

    }

}


// ////////////////////////////////////////
// HERE FUNCTIONS EXPOSED TO C/C++ FOLLOW
// ///////////////////////////////////////


void control_magnus(hipComplex* coeff_in, hipComplex *coeff_out, unsigned int amps, unsigned int n, float dt, unsigned int numSMs)
{
    dim3 threadsPerBlock(256/(amps*amps), amps,amps);
    dim3 numBlocks(n / threadsPerBlock.x+1, 1);
    //printf("n=%d und amps=%d \n",n,amps);
    //printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z, numBlocks.x, numBlocks.y, numBlocks.z);
    generate_magnus<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n, dt);
}



void control_midpoint(hipComplex* coeff_in, hipComplex *coeff_out, unsigned int amps, unsigned int n, unsigned int numSMs)
{
    dim3 threadsPerBlock(256/amps, amps);
    dim3 numBlocks(n / threadsPerBlock.x+1, 1);
    //printf("n=%d und amps=%d \n",n,amps);
    //printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z, numBlocks.x, numBlocks.y, numBlocks.z);
    generate_midpoint<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n);
}


void control_simpson(hipComplex* coeff_in, hipComplex *coeff_out, unsigned int amps, unsigned int n, unsigned int numSMs)
{
    dim3 threadsPerBlock(256/amps, amps);
    dim3 numBlocks((n-3)/2 / threadsPerBlock.x+1, 1);
    //printf("n=%d und amps=%d \n",n,amps);
    //printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z, numBlocks.x, numBlocks.y, numBlocks.z);
    generate_simpson<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n);
}