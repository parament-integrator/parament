#include "hip/hip_runtime.h"
/* Copyright 2021 Konstantin Herb, Pol Welter. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/


#include <hipblas.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#define NO_CUDA_STUBS
#include "control_expansion.h"

/*
 * Kernel for Magnus expansion of coefficient arrays (FP32)
 */
// 3D thread block indexing
__global__ void generate_magnus(hipComplex *coeffs_in, hipComplex *coeffs_out, int amps, int n, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Amp index
    int k = blockIdx.z * blockDim.z + threadIdx.z;  // Second amp index
    int len_new = (n-3)/2+1;
    hipComplex four = make_hipComplex(4,0);
    hipComplex six = make_hipComplex(6,0);

    const hipComplex commutatorFactor = make_hipComplex(0, dt/12.0);
    
    if (i < (n-3)/2+1 && j < amps && k == 0){
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_in[2*i+j*n], hipCmulf(four, coeffs_in[2*i+j*n+1]));
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_out[i+j*len_new], coeffs_in[2*i+j*n+2]);
        coeffs_out[i+j*len_new] = hipCdivf(coeffs_out[i+j*len_new], six);

        // Coefficients for commutator [H0,control H]
        int idx_comm = i+(j+amps)*len_new;
        coeffs_out[idx_comm] = hipCsubf(coeffs_in[2*i+j*n+2], coeffs_in[2*i+j*n]);
        coeffs_out[idx_comm] = hipCmulf(coeffs_out[idx_comm], commutatorFactor);
    }
    
    // Coefficient calculations for pairwise commutators of control Hamiltonians
    if (i < len_new && j < k && k < amps){
        int idx_old_amp1 = 2*i+n*j;
        int idx_old_amp2 = 2*i+n*k;
        int idx_new_pair = i+j*len_new+(k-1)*len_new + len_new*amps*2;
        //coeff_out[idx_new_pair] = coeff_in[idx_old_amp1]*coeff_in[idx_old_amp2+2]-coeff_in[idx_old_amp1+2]*coeff_in[idx_old_amp2];
        coeffs_out[idx_new_pair] = hipCmulf(coeffs_in[idx_old_amp1], coeffs_in[idx_old_amp2+2]);
        coeffs_out[idx_new_pair] = hipCsubf(coeffs_out[idx_new_pair],
                                        hipCmulf(coeffs_in[idx_old_amp1+2], coeffs_in[idx_old_amp2]));
        coeffs_out[idx_new_pair] = hipCmulf(coeffs_out[idx_new_pair], commutatorFactor);
    }
}

/*
 * Kernel for Magnus expansion of coefficient arrays (FP64)
 */
__global__ void generate_magnus_fp64(hipDoubleComplex *coeffs_in, hipDoubleComplex *coeffs_out, int amps, int n,
                                     double dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Amp index
    int k = blockIdx.z * blockDim.z + threadIdx.z;  // Second amp index
    int len_new = (n-3)/2+1;
    hipDoubleComplex four = make_hipDoubleComplex(4,0);
    hipDoubleComplex six = make_hipDoubleComplex(6,0);

    const hipDoubleComplex commutatorFactor = make_hipDoubleComplex(0, dt/12.0);
    
    if (i < (n-3)/2+1 && j < amps && k == 0){
        coeffs_out[i+j*len_new] = hipCadd(coeffs_in[2*i+j*n], hipCmul(four,coeffs_in[2*i+j*n+1]));
        coeffs_out[i+j*len_new] = hipCadd(coeffs_out[i+j*len_new], coeffs_in[2*i+j*n+2]);
        coeffs_out[i+j*len_new] = hipCdiv(coeffs_out[i+j*len_new], six);

        // Coefficients for commutator [H0,control H]
        int idx_comm = i+(j+amps)*len_new;
        coeffs_out[idx_comm] = hipCsub(coeffs_in[2*i+j*n+2], coeffs_in[2*i+j*n]);
        coeffs_out[idx_comm] = hipCmul(coeffs_out[idx_comm], commutatorFactor);
    }
    
    // Coefficient calculations for pairwise commutators of control Hamiltonians
    if (i < len_new && j < k && k < amps){
        int idx_old_amp1 = 2*i+n*j;
        int idx_old_amp2 = 2*i+n*k;
        int idx_new_pair = i+j*len_new+(k-1)*len_new + len_new*amps*2;
        //coeff_out[idx_new_pair] = coeff_in[idx_old_amp1]*coeff_in[idx_old_amp2+2]-coeff_in[idx_old_amp1+2]*coeff_in[idx_old_amp2];
        coeffs_out[idx_new_pair] = hipCmul(coeffs_in[idx_old_amp1], coeffs_in[idx_old_amp2+2]);
        coeffs_out[idx_new_pair] = hipCsub(coeffs_out[idx_new_pair],
                                        hipCmul(coeffs_in[idx_old_amp1+2], coeffs_in[idx_old_amp2]));
        coeffs_out[idx_new_pair] = hipCmul(coeffs_out[idx_new_pair], commutatorFactor);
    }
}

/*
 * Kernel for applying midpoint rule to coefficient arrays (FP32)
 */
// 2D thread block indexing
__global__ void generate_midpoint(hipComplex *coeffs_in, hipComplex *coeffs_out, int amps, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Control field index 
    hipComplex half = make_hipComplex(0.5,0);

    if (i < n - 1 && j < amps){
        coeffs_out[i+j*(n-1)] = hipCaddf(coeffs_in[i+j*n],coeffs_in[i+j*n+1]);
        coeffs_out[i+j*(n-1)] = hipCmulf(half,coeffs_out[i+j*(n-1)]);
    }
}

/*
 * Kernel for applying midpoint rule to coefficient arrays (FP64)
 */
// 2D thread block indexing
__global__ void generate_midpoint_fp64(hipDoubleComplex *coeffs_in, hipDoubleComplex *coeffs_out, int amps, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Control field index 
    hipDoubleComplex half = make_hipDoubleComplex(0.5,0);

    if (i < n - 1 && j < amps){
        coeffs_out[i+j*(n-1)] = hipCadd(coeffs_in[i+j*n],coeffs_in[i+j*n+1]);
        coeffs_out[i+j*(n-1)] = hipCmul(half,coeffs_out[i+j*(n-1)]);
    }
}

/*
 * Kernel for applying Simpson's rule to coefficient arrays (FP32)
 */
// 2D thread block indexing
__global__ void generate_simpson(hipComplex *coeffs_in, hipComplex *coeffs_out, int amps, int n)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Control field index 
    hipComplex four = make_hipComplex(4,0);
    hipComplex six = make_hipComplex(6,0);
    int len_new = (n-3)/2+1;

    if (i < (n-3)/2+1 && j < amps){
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_in[2*i+j*n], hipCmulf(four,coeffs_in[2*i+j*n+1]));
        coeffs_out[i+j*len_new] = hipCaddf(coeffs_out[i+j*len_new], coeffs_in[2*i+j*n+2]);
        coeffs_out[i+j*len_new] = hipCdivf(coeffs_out[i+j*len_new], six);
    }
}

/*
 * Kernel for applying Simpson's rule to coefficient arrays (FP64)
 */
__global__ void generate_simpson_fp64(hipDoubleComplex *coeffs_in, hipDoubleComplex *coeffs_out, int amps, int n)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // Timepoint index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Control field index 
    hipDoubleComplex four = make_hipDoubleComplex(4,0);
    hipDoubleComplex six = make_hipDoubleComplex(6,0);
    int len_new = (n-3)/2+1;

    if (i < (n-3)/2+1 && j < amps){
        coeffs_out[i+j*len_new] = hipCadd(coeffs_in[2*i+j*n], hipCmul(four,coeffs_in[2*i+j*n+1]));
        coeffs_out[i+j*len_new] = hipCadd(coeffs_out[i+j*len_new], coeffs_in[2*i+j*n+2]);
        coeffs_out[i+j*len_new] = hipCdiv(coeffs_out[i+j*len_new], six);
    }
}


// ////////////////////////////////////////
// HERE FUNCTIONS EXPOSED TO C/C++ FOLLOW
// ///////////////////////////////////////
void control_magnus(hipComplex* coeff_in, hipComplex *coeff_out, unsigned int amps, unsigned int n, float dt,
                    unsigned int numSMs)
{
    dim3 threadsPerBlock(256/(amps*amps), amps,amps);
    dim3 numBlocks(n / threadsPerBlock.x+1, 1);
    assert(hipPeekAtLastError() == hipSuccess);
    generate_magnus<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n, dt);
    assert(hipPeekAtLastError() == hipSuccess);
}

void control_magnus(hipDoubleComplex* coeff_in, hipDoubleComplex *coeff_out, unsigned int amps, unsigned int n,
                    double dt, unsigned int numSMs)
{
    dim3 threadsPerBlock(256/(amps*amps), amps,amps);
    dim3 numBlocks(n / threadsPerBlock.x+1, 1);
    assert(hipPeekAtLastError() == hipSuccess);
    generate_magnus_fp64<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n, dt);
    assert(hipPeekAtLastError() == hipSuccess);
}


void control_midpoint(hipComplex* coeff_in, hipComplex *coeff_out, unsigned int amps, unsigned int n, unsigned int numSMs)
{
    dim3 threadsPerBlock(256/amps, amps);
    dim3 numBlocks(n / threadsPerBlock.x+1, 1);
    assert(hipPeekAtLastError() == hipSuccess);
    generate_midpoint<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n);
    assert(hipPeekAtLastError() == hipSuccess);
}

void control_midpoint(hipDoubleComplex* coeff_in, hipDoubleComplex *coeff_out, unsigned int amps, unsigned int n,
                      unsigned int numSMs)
{
    dim3 threadsPerBlock(256/amps, amps);
    dim3 numBlocks(n / threadsPerBlock.x+1, 1);
    assert(hipPeekAtLastError() == hipSuccess);
    generate_midpoint_fp64<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n);
    assert(hipPeekAtLastError() == hipSuccess);
}


void control_simpson(hipComplex* coeff_in, hipComplex *coeff_out, unsigned int amps, unsigned int n, unsigned int numSMs)
{
    dim3 threadsPerBlock(256/amps, amps);
    dim3 numBlocks((n-3)/2 / threadsPerBlock.x+1, 1);
    assert(hipPeekAtLastError() == hipSuccess);
    generate_simpson<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n);
    assert(hipPeekAtLastError() == hipSuccess);
}

void control_simpson(hipDoubleComplex* coeff_in, hipDoubleComplex *coeff_out, unsigned int amps, unsigned int n,
                     unsigned int numSMs)
{
    dim3 threadsPerBlock(256/amps, amps);
    dim3 numBlocks((n-3)/2 / threadsPerBlock.x+1, 1);
    assert(hipPeekAtLastError() == hipSuccess);
    generate_simpson_fp64<<<numBlocks, threadsPerBlock>>>(coeff_in, coeff_out, amps, n);
    assert(hipPeekAtLastError() == hipSuccess);
}
